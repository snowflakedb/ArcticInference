#include "hip/hip_runtime.h"
#include "custom_ops.h"
#include "dispatch_utils.h"
#include "quant_utils.cuh"
#include "vectorization_utils.cuh"

#include <torch/hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_fp8.h>

#include <vector>

namespace vllm {

// Used to copy/convert one element
template <typename OutT, typename InT, Fp8KVCacheDataType kv_dt>
struct CopyWithScaleOp {
  float scale;

  __device__ __forceinline__ void operator()(OutT& dst, const InT src) const {
    if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
      dst = static_cast<OutT>(src);
    } else {
      dst = fp8::scaled_convert<OutT, InT, kv_dt>(src, scale);
    }
  }
};

namespace nvfp4 {

constexpr int CVT_FP4_ELTS_PER_THREAD = 8;   // 8 input elts → 1 uint32
constexpr int CVT_FP4_SF_VEC_SIZE      = 16; // 1 scale per 16 elts

template <typename T> struct TypePair { using V2 = half2; };
template <> struct TypePair<half> { using V2 = half2; };
template <> struct TypePair<__hip_bfloat16> { using V2 = __hip_bfloat162; };

template <class T>
struct PackedVec {
  // 8 elements total, loaded as 4 x {half2 or bfloat162} (16B)
  typename TypePair<T>::V2 elts[4];
};

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900 && __CUDA_ARCH__ < 1000
__device__ inline uint8_t float_to_e2m1_rn(float val) {
  if (isnan(val)) return 0x0;
  if (isinf(val)) val = val < 0.f ? -6.f : 6.f;
  uint32_t sign_bit = (reinterpret_cast<uint32_t&>(val) & 0x80000000) >> 28;
  float x = fabsf(val);
  uint8_t mag;
  if (x > 5.0f)      mag = 0x7; // 6.0
  else if (x > 3.5f) mag = 0x6; // 4.0
  else if (x > 2.5f) mag = 0x5; // 3.0
  else if (x > 1.75f)mag = 0x4; // 2.0
  else if (x > 1.25f)mag = 0x3; // 1.5
  else if (x > 0.75f)mag = 0x2; // 1.0
  else if (x > 0.25f)mag = 0x1; // 0.5
  else               mag = 0x0; // 0.0
  return sign_bit | mag;
}
#endif

inline __device__ float rcp_approx_ftz(float a) {
  float b; asm volatile("rcp.approx.ftz.f32 %0, %1;" : "=f"(b) : "f"(a)); return b;
}

inline __device__ uint32_t fp32_8_to_e2m1x2_packed(float (&a)[8]) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  uint32_t val;
  asm volatile(
      "{.reg .b8 b0,b1,b2,b3;"
      "cvt.rn.satfinite.e2m1x2.f32 b0, %2, %1;"
      "cvt.rn.satfinite.e2m1x2.f32 b1, %4, %3;"
      "cvt.rn.satfinite.e2m1x2.f32 b2, %6, %5;"
      "cvt.rn.satfinite.e2m1x2.f32 b3, %8, %7;"
      "mov.b32 %0, {b0,b1,b2,b3};}"
      : "=r"(val)
      : "f"(a[0]), "f"(a[1]), "f"(a[2]), "f"(a[3]),
        "f"(a[4]), "f"(a[5]), "f"(a[6]), "f"(a[7]));
  return val;
#elif defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
  uint32_t r = 0; uint8_t* rb = reinterpret_cast<uint8_t*>(&r);
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    uint8_t v1 = float_to_e2m1_rn(a[2*i]);
    uint8_t v2 = float_to_e2m1_rn(a[2*i+1]);
    rb[i] = (v2 << 4) | (v1 & 0x0F);
  }
  return r;
#else
  return 0u;
#endif
}

inline __device__ uint32_t fp32_8_to_e2m1x2_packed(float2 (&a)[4]) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  uint32_t val;
  asm volatile(
      "{.reg .b8 b0,b1,b2,b3;"
      "cvt.rn.satfinite.e2m1x2.f32 b0, %2, %1;"
      "cvt.rn.satfinite.e2m1x2.f32 b1, %4, %3;"
      "cvt.rn.satfinite.e2m1x2.f32 b2, %6, %5;"
      "cvt.rn.satfinite.e2m1x2.f32 b3, %8, %7;"
      "mov.b32 %0, {b0,b1,b2,b3};}"
      : "=r"(val)
      : "f"(a[0].x), "f"(a[0].y), "f"(a[1].x), "f"(a[1].y),
        "f"(a[2].x), "f"(a[2].y), "f"(a[3].x), "f"(a[3].y));
  return val;
#elif defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
  uint32_t r = 0; uint8_t* rb = reinterpret_cast<uint8_t*>(&r);
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    uint8_t v1 = float_to_e2m1_rn(a[i].x);
    uint8_t v2 = float_to_e2m1_rn(a[i].y);
    rb[i] = (v2 << 4) | (v1 & 0x0F);
  }
  return r;
#else
  return 0u;
#endif
}

// Per-thread (8 elts) → one packed uint32; pairs of threads share the SF.
template <class T, bool UE8M0_SF=false>
__device__ uint32_t warp_quant_8_to_nvfp4(PackedVec<T>& vec8,
                                          float sf_scale_val,
                                          uint8_t* sf_out_byte_if_leader) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
  // 1) gather local abs max across 8 values
  auto localMax = __habs2(vec8.elts[0]);
#pragma unroll
  for (int i = 1; i < 4; ++i) { // 4 half2 chunks = 8 elts
    localMax = __hmax2(localMax, __habs2(vec8.elts[i]));
  }
  // 2) partner thread (lane^1) to cover 16 elts
  localMax = __hmax2(__shfl_xor_sync(0xffffffff, localMax, 1), localMax);
  float vmax = float(__hmax(localMax.x, localMax.y));

  // 3) SF = sf_scale_val * (vmax / 6.0)
  float sf_val = sf_scale_val * (vmax * rcp_approx_ftz(6.0f));
  uint8_t sf_byte;
  if constexpr (UE8M0_SF) {
    uint32_t tmp = reinterpret_cast<uint32_t&>(sf_val) >> 23;
    sf_byte = tmp & 0xff;
    reinterpret_cast<uint32_t&>(sf_val) = tmp << 23;
  } else {
    __hip_fp8_e4m3_fnuz tmp = __hip_fp8_e4m3_fnuz(sf_val);
    reinterpret_cast<__hip_fp8_e4m3_fnuz&>(sf_byte) = tmp;
    sf_val = float(tmp);
  }

  float out_scale = (sf_val != 0.f)
                    ? rcp_approx_ftz(sf_val * rcp_approx_ftz(sf_scale_val))
                    : 0.0f;

  if (sf_out_byte_if_leader) { *sf_out_byte_if_leader = sf_byte; }

  // 4) scale & pack 8 values
  float2 f2[4];
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    if constexpr (std::is_same_v<T, half>) {
      f2[i] = __half22float2(vec8.elts[i]);
    } else {
      f2[i] = __bfloat1622float2(vec8.elts[i]);
    }
    f2[i].x *= out_scale; f2[i].y *= out_scale;
  }
  return fp32_8_to_e2m1x2_packed(f2);
#else
  return 0u;
#endif
}

// Utility: compute per-head packing sizes
__host__ __device__ inline int q_words_per_head(int head_size) {
  return head_size / 8;
}
__host__ __device__ inline int sf_bytes_per_head(int head_size) {
  int scales = head_size / 16;                 // 1 byte / 16 elts
  int pad4   = ((scales + 3) / 4) * 4;         // pad to 4 bytes
  return pad4;
}
__host__ __device__ inline int head_words_total(int head_size) {
  return q_words_per_head(head_size) + sf_bytes_per_head(head_size)/4;
}

} // namespace nvfp4

// NVFP4 cache writer: converts FP16/BF16 key/value to NVFP4 and writes
// packed data + per-16 scale bytes into [key_cache|value_cache].
template <typename scalar_t, bool UE8M0_SF=false>
__global__ void reshape_and_cache_flash_nvfp4_kernel(
    const scalar_t* __restrict__ key,    // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,  // [num_tokens, num_heads, head_size]
    uint32_t* __restrict__ key_cache,    // int32 view; row=[Q words][SF bytes/4]
    uint32_t* __restrict__ value_cache,
    const int64_t* __restrict__ slot_mapping,       // [num_tokens or actual]
    const int64_t block_stride_words, const int64_t page_stride_words,
    const int64_t head_stride_words, const int64_t key_stride_elems,
    const int64_t value_stride_elems, const int num_heads, const int head_size,
    const int block_size, const float* k_sf_scale_ptr, const float* v_sf_scale_ptr) {

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx  = slot_mapping[token_idx];
  if (slot_idx < 0) return;

  const int64_t block_idx   = slot_idx / block_size;
  const int64_t block_off   = slot_idx % block_size;

  // Source row bases for this token
  const scalar_t* __restrict__ key_src   = key   + token_idx * key_stride_elems;
  const scalar_t* __restrict__ value_src = value + token_idx * value_stride_elems;

  // Destination base (int32 words) for this token
  uint32_t* __restrict__ key_dst_words =
      key_cache + block_idx * block_stride_words + block_off * page_stride_words;
  uint32_t* __restrict__ value_dst_words =
      value_cache + block_idx * block_stride_words + block_off * page_stride_words;

  const float k_sf_scale = (k_sf_scale_ptr ? *k_sf_scale_ptr : 1.0f);
  const float v_sf_scale = (v_sf_scale_ptr ? *v_sf_scale_ptr : 1.0f);

  const int q_words = nvfp4::q_words_per_head(head_size);
  // NOTE: sf_bytes is padded to 4, so we can safely advance the words pointer
  // to the beginning of the SF region as (base + q_words).
  const int head_words = nvfp4::head_words_total(head_size);

  const int lane = threadIdx.x & 31;         // 0..31
  const int warp_id = threadIdx.x >> 5;      // warp within block
  const int warps_per_block = blockDim.x >> 5;

  // Each warp iterates over heads assigned to it.
  for (int head = warp_id; head < num_heads; head += warps_per_block) {
    // Per-head contiguous input
    const scalar_t* __restrict__ k_src_h = key_src   + head * head_size;
    const scalar_t* __restrict__ v_src_h = value_src + head * head_size;

    // Per-head output slices (int32 words)
    uint32_t* __restrict__ k_head_base = key_dst_words   + static_cast<int64_t>(head) * head_stride_words;
    uint32_t* __restrict__ v_head_base = value_dst_words + static_cast<int64_t>(head) * head_stride_words;

    uint32_t* __restrict__ k_q_out = k_head_base;                     // Q words region
    uint8_t*  __restrict__ k_sf_out = reinterpret_cast<uint8_t*>(k_head_base + q_words); // SF bytes region

    uint32_t* __restrict__ v_q_out = v_head_base;
    uint8_t*  __restrict__ v_sf_out = reinterpret_cast<uint8_t*>(v_head_base + q_words);

    // Number of 8‑elt groups in this head
    const int n_groups8 = head_size / nvfp4::CVT_FP4_ELTS_PER_THREAD;

    // Thread‑strided over 8‑elt groups; pairs of lanes (0/1, 2/3, …) share a scale
    for (int col = lane; col < n_groups8; col += 32) {
      // Load 8 contiguous elements as a 16‑byte packed vector
      using PV = nvfp4::PackedVec<scalar_t>;
      const PV* __restrict__ k_src8 = reinterpret_cast<const PV*>(k_src_h);
      const PV* __restrict__ v_src8 = reinterpret_cast<const PV*>(v_src_h);
      PV k_vec8 = k_src8[col];
      PV v_vec8 = v_src8[col];

      // Only the even lane of each pair writes the scale byte (one per 16 elts)
      uint8_t* k_sf_ptr = ((lane & 1) == 0) ? (k_sf_out + (col >> 1)) : nullptr;
      uint8_t* v_sf_ptr = ((lane & 1) == 0) ? (v_sf_out + (col >> 1)) : nullptr;

      uint32_t k_packed =
          nvfp4::warp_quant_8_to_nvfp4<scalar_t, UE8M0_SF>(k_vec8, k_sf_scale, k_sf_ptr);
      uint32_t v_packed =
          nvfp4::warp_quant_8_to_nvfp4<scalar_t, UE8M0_SF>(v_vec8, v_sf_scale, v_sf_ptr);

      k_q_out[col] = k_packed;
      v_q_out[col] = v_packed;
    }
  }
#else
  // Compile-time guard: NVFP4 path requires SM90+ for performant implementation
  if (threadIdx.x == 0 && blockIdx.x == 0) { /* no-op on older arch */ }
#endif
}

} // namespace vllm

void reshape_and_cache_flash(
    torch::Tensor& key,        // [num_tokens, num_heads, head_size]
    torch::Tensor& value,      // [num_tokens, num_heads, head_size]
    torch::Tensor& key_cache,  // NVFP4: int32 view with packed [Q|S]
    torch::Tensor& value_cache,
    torch::Tensor& slot_mapping,
    const std::string& kv_cache_dtype,
    torch::Tensor& k_scale,
    torch::Tensor& v_scale) {

  int num_tokens = slot_mapping.size(0);
  int num_heads  = key.size(1);
  int head_size  = key.size(2);
  int block_size = key_cache.size(1);

  int64_t key_stride   = key.stride(0);
  int64_t value_stride = value.stride(0);

  int64_t block_stride = key_cache.stride(0);
  int64_t page_stride  = key_cache.stride(1);
  int64_t head_stride  = key_cache.stride(2);
  TORCH_CHECK(key_cache.stride(0) == value_cache.stride(0));

  dim3 grid(num_tokens);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(key));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(head_size % 16 == 0,
              "nvfp4: head_size must be a multiple of 16.");
  TORCH_CHECK(key_cache.scalar_type() == torch::kInt32 &&
              value_cache.scalar_type() == torch::kInt32,
              "nvfp4: key_cache/value_cache must be int32 tensors storing "
              "packed FP4 data followed by scale bytes.");

  // Block size: choose a multiple of 32; 512 works well in practice.
  dim3 block(std::min(512, std::max(32, num_heads * 32)));

  // Launch per input type
  switch (key.scalar_type()) {
    case torch::kHalf: {
      vllm::reshape_and_cache_flash_nvfp4_kernel<half, false>
          <<<grid, block, 0, stream>>>(
              reinterpret_cast<half const*>(key.data_ptr()),
              reinterpret_cast<half const*>(value.data_ptr()),
              reinterpret_cast<uint32_t*>(key_cache.data_ptr()),
              reinterpret_cast<uint32_t*>(value_cache.data_ptr()),
              slot_mapping.data_ptr<int64_t>(),
              block_stride, page_stride, head_stride,
              key_stride, value_stride,
              num_heads, head_size, block_size,
              reinterpret_cast<const float*>(k_scale.data_ptr()),
              reinterpret_cast<const float*>(v_scale.data_ptr()));
      break;
    }
    case torch::kBFloat16: {
      vllm::reshape_and_cache_flash_nvfp4_kernel<__hip_bfloat16, false>
          <<<grid, block, 0, stream>>>(
              reinterpret_cast<__hip_bfloat16 const*>(key.data_ptr()),
              reinterpret_cast<__hip_bfloat16 const*>(value.data_ptr()),
              reinterpret_cast<uint32_t*>(key_cache.data_ptr()),
              reinterpret_cast<uint32_t*>(value_cache.data_ptr()),
              slot_mapping.data_ptr<int64_t>(),
              block_stride, page_stride, head_stride,
              key_stride, value_stride,
              num_heads, head_size, block_size,
              reinterpret_cast<const float*>(k_scale.data_ptr()),
              reinterpret_cast<const float*>(v_scale.data_ptr()));
      break;
    }
    default:
      TORCH_CHECK(false, "nvfp4: unsupported key/value dtype (expected FP16/BF16).");
  }
}
