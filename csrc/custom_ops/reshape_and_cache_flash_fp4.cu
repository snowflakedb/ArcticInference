#include "hip/hip_runtime.h"
#include "custom_ops.h"
#include "dispatch_utils.h"
#include "quant_utils.cuh"
#include "vectorization_utils.cuh"

#include <torch/hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#include <vector>

namespace vllm {

// Used to copy/convert one element
template <typename OutT, typename InT, Fp8KVCacheDataType kv_dt>
struct CopyWithScaleOp {
  float scale;

  __device__ __forceinline__ void operator()(OutT& dst, const InT src) const {
    if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
      dst = static_cast<OutT>(src);
    } else {
      dst = fp8::scaled_convert<OutT, InT, kv_dt>(src, scale);
    }
  }
};

template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key,    // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,  // [num_tokens, num_heads, head_size]
    cache_t* __restrict__ key_cache,     // NHD or HND, shape see comments below
    cache_t* __restrict__ value_cache,   // same above
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int64_t block_stride, const int64_t page_stride,
    const int64_t head_stride, const int64_t key_stride,
    const int64_t value_stride, const int num_heads, const int head_size,
    const int block_size, const float* k_scale, const float* v_scale) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  // NOTE: slot_idx can be -1 if the token is padded
  if (slot_idx < 0) {
    return;
  }
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;
  const int n_elems = num_heads * head_size;

  // pointers to the beginning of the source row for this token.
  const scalar_t* __restrict__ key_src = key + token_idx * key_stride;
  const scalar_t* __restrict__ value_src = value + token_idx * value_stride;

  // find the start position inside the kv-cache for this token.
  cache_t* __restrict__ key_dst =
      key_cache + block_idx * block_stride + block_offset * page_stride;
  cache_t* __restrict__ value_dst =
      value_cache + block_idx * block_stride + block_offset * page_stride;

  // this is true for the NHD layout where `head_stride == head_size`
  const bool is_contiguous_heads = (head_stride == head_size);

  float k_scale_val = (kv_dt == Fp8KVCacheDataType::kAuto) ? 0.f : *k_scale;
  float v_scale_val = (kv_dt == Fp8KVCacheDataType::kAuto) ? 0.f : *v_scale;
  constexpr int VEC_SIZE = (sizeof(scalar_t) == 2) ? 8 : 4;
  CopyWithScaleOp<cache_t, scalar_t, kv_dt> k_op{k_scale_val};
  CopyWithScaleOp<cache_t, scalar_t, kv_dt> v_op{v_scale_val};
  if (is_contiguous_heads) {
    // NHD layout
    // kv cache: [num_blocks, block_size, num_heads, head_size]
    vectorize_with_alignment<VEC_SIZE>(key_src, key_dst, n_elems, threadIdx.x,
                                       blockDim.x, k_op);

    vectorize_with_alignment<VEC_SIZE>(value_src, value_dst, n_elems,
                                       threadIdx.x, blockDim.x, v_op);

  } else {
    // HND layout: heads are strided, but each head_size segment is contiguous
    // kv cache: [num_blocks, num_heads, block_size, head_size]
    const int lane = threadIdx.x & 31;     // 0..31 within warp
    const int warp_id = threadIdx.x >> 5;  // warp index within block
    const int warps_per_block = blockDim.x >> 5;

    for (int head = warp_id; head < num_heads; head += warps_per_block) {
      const scalar_t* __restrict__ k_src_h = key_src + head * head_size;
      const scalar_t* __restrict__ v_src_h = value_src + head * head_size;

      cache_t* __restrict__ k_dst_h =
          key_dst + static_cast<int64_t>(head) * head_stride;
      cache_t* __restrict__ v_dst_h =
          value_dst + static_cast<int64_t>(head) * head_stride;

      // within each head, let the 32 threads of the warp perform the vector
      // copy
      vectorize_with_alignment<VEC_SIZE>(k_src_h, k_dst_h, head_size, lane, 32,
                                         k_op);

      vectorize_with_alignment<VEC_SIZE>(v_src_h, v_dst_h, head_size, lane, 32,
                                         v_op);
    }
  }
}

} // namespace vllm

#define CALL_RESHAPE_AND_CACHE_FLASH(KV_T, CACHE_T, KV_DTYPE)             \
  vllm::reshape_and_cache_flash_kernel<KV_T, CACHE_T, KV_DTYPE>           \
      <<<grid, block, 0, stream>>>(                                       \
          reinterpret_cast<KV_T*>(key.data_ptr()),                        \
          reinterpret_cast<KV_T*>(value.data_ptr()),                      \
          reinterpret_cast<CACHE_T*>(key_cache.data_ptr()),               \
          reinterpret_cast<CACHE_T*>(value_cache.data_ptr()),             \
          slot_mapping.data_ptr<int64_t>(), block_stride, page_stride,    \
          head_stride, key_stride, value_stride, num_heads, head_size,    \
          block_size, reinterpret_cast<const float*>(k_scale.data_ptr()), \
          reinterpret_cast<const float*>(v_scale.data_ptr()));

void reshape_and_cache_flash(
    torch::Tensor& key,        // [num_tokens, num_heads, head_size]
    torch::Tensor& value,      // [num_tokens, num_heads, head_size]
    torch::Tensor& key_cache,  // [num_blocks, block_size, num_heads, head_size]
    torch::Tensor&
        value_cache,  // [num_blocks, block_size, num_heads, head_size]
    torch::Tensor& slot_mapping,  // [num_tokens] or [num_actual_tokens]
    const std::string& kv_cache_dtype, torch::Tensor& k_scale,
    torch::Tensor& v_scale) {
  // NOTE(woosuk): In vLLM V1, key.size(0) can be different from
  // slot_mapping.size(0) because of padding for CUDA graphs.
  // In vLLM V0, key.size(0) is always equal to slot_mapping.size(0) because
  // both include padding.
  // In vLLM V1, however, key.size(0) can be larger than slot_mapping.size(0)
  // since key includes padding for CUDA graphs, while slot_mapping does not.
  // In this case, slot_mapping.size(0) represents the actual number of tokens
  // before padding.
  // For compatibility with both cases, we use slot_mapping.size(0) as the
  // number of tokens.
  int num_tokens = slot_mapping.size(0);
  int num_heads = key.size(1);
  int head_size = key.size(2);
  int block_size = key_cache.size(1);

  int64_t key_stride = key.stride(0);
  int64_t value_stride = value.stride(0);
  int64_t block_stride = key_cache.stride(0);
  int64_t page_stride = key_cache.stride(1);
  int64_t head_stride = key_cache.stride(2);
  TORCH_CHECK(key_cache.stride(0) == value_cache.stride(0));

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(key));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_BY_KV_CACHE_DTYPE(key.dtype(), kv_cache_dtype,
                             CALL_RESHAPE_AND_CACHE_FLASH);
}